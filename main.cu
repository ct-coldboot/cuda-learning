/****
 * main.cu
 *
 * Classic hello world with CUDA flavor
 **/

#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        return 1;
    }

    if (deviceCount == 0) {
        std::cout << "No CUDA-enabled devices found." << std::endl;
    } else {
        std::cout << "Found " << deviceCount << " CUDA device(s)." << std::endl;
        
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0); // Get properties for device 0

        std::cout << "Device 0: " << prop.name << std::endl;
        std::cout << "Compute Capability: " << prop.major << "." << prop.minor << std::endl;
    }

    return 0;
}